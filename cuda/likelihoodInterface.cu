#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
//
//    Copyright 2010
//
//    This file is part of rootpwa
//
//    rootpwa is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    rootpwa is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with rootpwa. If not, see <http://www.gnu.org/licenses/>.
//
///////////////////////////////////////////////////////////////////////////
//-------------------------------------------------------------------------
// File and Version Information:
// $Rev::                             $: revision of last commit
// $Author::                          $: author of last commit
// $Date::                            $: date of last commit
//
// Description:
//      interface for CUDA likelihood functions
//
//
// Author List:
//      Philipp Meyer          TUM            (original author)
//
//
//-------------------------------------------------------------------------


#include <hip/hip_runtime.h>
#include <cutil_inline.h>

#include "nDimArrayUtils.hpp"
#include "reportingUtils.hpp"

#include "complex.cuh"
#include "likelihoodInterface.cuh"
#include "likelihoodKernel.cu"


using namespace std;
using namespace rpwa;
using namespace rpwa::cuda;


template<typename complexT> likelihoodInterface<complexT> likelihoodInterface<complexT>::_instance;

template<typename complexT> bool           likelihoodInterface<complexT>::_cudaInitialized    = false;
template<typename complexT> int            likelihoodInterface<complexT>::_nmbOfCudaDevices   = 0;
template<typename complexT> int            likelihoodInterface<complexT>::_cudaDeviceId       = -1;
template<typename complexT> hipDeviceProp_t likelihoodInterface<complexT>::_cudaDeviceProp;
template<typename complexT> unsigned int   likelihoodInterface<complexT>::_nmbBlocks          = 0;
template<typename complexT> unsigned int   likelihoodInterface<complexT>::_nmbThreadsPerBlock = 0;
template<typename complexT> complexT*      likelihoodInterface<complexT>::_d_decayAmps        = 0;
template<typename complexT> unsigned int   likelihoodInterface<complexT>::_nmbEvents          = 0;
template<typename complexT> unsigned int   likelihoodInterface<complexT>::_nmbWavesRefl[2]    = {0, 0};
template<typename complexT> bool           likelihoodInterface<complexT>::_debug              = false;


template<typename complexT>
likelihoodInterface<complexT>::~likelihoodInterface()
{
	if (_d_decayAmps) {
		cutilSafeCall(hipFree(_d_decayAmps));
		cutilSafeCall(hipDeviceReset());
	}
}


template<typename complexT>
unsigned int
likelihoodInterface<complexT>::totalDeviceMem()
{
	if (not _cudaInitialized) {
		printWarn << "cannot estimate total device memory. CUDA device not initialized." << endl;
		return 0;
	}
	return _cudaDeviceProp.totalGlobalMem;
}


template<typename complexT>
unsigned int
likelihoodInterface<complexT>::availableDeviceMem()
{
	// unfortunately the runtime API does not provide a call to get the
	// available memory on the device so we have to do it via the driver
	// API; sigh
	if (not _cudaInitialized) {
		printWarn << "cannot estimate available device memory. CUDA device not initialized." << endl;
		return 0;
	}
	if (hipInit(0) != hipSuccess) {
		printWarn << "hipInit() failed." << endl;
		return 0;
	}
  hipDevice_t device;
  if (hipDeviceGet(&device, _cudaDeviceId) != hipSuccess) {
	  printWarn << "hipDeviceGet() failed." << endl;
		return 0;
	}
  hipCtx_t context;
  if (hipCtxCreate(&context, 0, device) != hipSuccess) {
	  printWarn << "hipCtxCreate() failed." << endl;
		return 0;
	}
	unsigned int free, total;
	if (hipMemGetInfo(&free, &total) != hipSuccess) {
	  printWarn << "hipMemGetInfo() failed." << endl;
		return 0;
	}
  if (cuCtxDetach(context) != hipSuccess)
	  printWarn << "cuCtxDetach() failed." << endl;
  return free;
}


template<typename complexT>
const struct hipDeviceProp_t*
likelihoodInterface<complexT>::deviceProperties()
{
	if (not _cudaInitialized)
		return 0;
	return &_cudaDeviceProp;
}


template<typename complexT>
bool
likelihoodInterface<complexT>::init
(const complexT*    decayAmps,        // array of decay amplitudes; [iRefl][iWave][iEvt] or [iEvt][iRefl][iWave]
 const unsigned int nmbDecayAmps,     // number of elements in decay amplitude array
 const unsigned int nmbEvents,        // total number of events
 const unsigned int nmbWavesRefl[2],  // number of waves for each reflectivity
 const bool         reshuffleArray)   // if set devcay amplitude array is reshuffled from [iEvt][iRefl][iWave] to [iRefl][iWave][iEvt]
{
	if (not initCudaDevice()) {
		printWarn << "problems initializing CUDA device" << endl;
		return false;
	}
	if (_debug)
		printInfo << _instance;
	if (not loadDecayAmps(decayAmps, nmbDecayAmps, nmbEvents, nmbWavesRefl, reshuffleArray)) {
		printWarn << "problems loading decay amplitudes into CUDA device" << endl;
		return false;
	}
	return true;
}


template<typename complexT>
bool
likelihoodInterface<complexT>::initCudaDevice()
{
	_cudaInitialized = false;

  // get number of CUDA devices in system
  cutilSafeCall(hipGetDeviceCount(&_nmbOfCudaDevices));
  if (_nmbOfCudaDevices == 0) {
    printWarn << "there are no CUDA devices in the system" << endl;
    return false;
  }
  printInfo << "found " << _nmbOfCudaDevices << " CUDA device(s)" << endl;

  // use most powerful GPU in system
  _cudaDeviceId = cutGetMaxGflopsDeviceId();
  cutilSafeCall(hipGetDeviceProperties(&_cudaDeviceProp, _cudaDeviceId));
  printInfo << "using CUDA device[" << _cudaDeviceId << "]: '" << _cudaDeviceProp.name << "'" << endl;
  // fields for both major & minor fields are 9999, if device is not present
  if ((_cudaDeviceProp.major == 9999) and (_cudaDeviceProp.minor == 9999)) {
	  printWarn << "there is no CUDA device with ID " << _cudaDeviceId << endl;
	  return false;
  }
  cutilSafeCall(hipSetDevice(_cudaDeviceId));

  // setup thread grid paramters
	_nmbBlocks          = _cudaDeviceProp.multiProcessorCount;
	//nmbThreadsPerBlock = _cudaDeviceProp.maxThreadsPerBlock;
	_nmbThreadsPerBlock = 448;
	printInfo << "using " << _nmbBlocks << " x " << _nmbThreadsPerBlock << " = "
	          << _nmbBlocks * _nmbThreadsPerBlock << " CUDA threads for likelihood calculation" << endl;

	_cudaInitialized = true;
	return true;
}


template<typename complexT>
bool
likelihoodInterface<complexT>::loadDecayAmps
(const complexT*    decayAmps,        // array of decay amplitudes; [iRefl][iWave][iEvt] or [iEvt][iRefl][iWave]
 const unsigned int nmbDecayAmps,     // number of elements in decay amplitude array
 const unsigned int nmbEvents,        // total number of events
 const unsigned int nmbWavesRefl[2],  // number of waves for each reflectivity
 const bool         reshuffleArray)   // if set devcay amplitude array is reshuffled from [iEvt][iRefl][iWave] to [iRefl][iWave][iEvt]
{
	if (not decayAmps) {
		printErr << "null pointer to decay amplitudes. aborting." << endl;
		throw;
	}
	if (not _cudaInitialized) {
		printWarn << "cannot load decay amplitudes. CUDA device is not initialized." << endl;
		return false;
	}

	_nmbEvents       = nmbEvents;
	_nmbWavesRefl[0] = nmbWavesRefl[0];
	_nmbWavesRefl[1] = nmbWavesRefl[1];

	complexT* h_decayAmps = 0;
	if (reshuffleArray) {
		// change memory layout of decay amplitudes from [iEvt][iRefl][iWave] to [iRefl][iWave][iEvt]
		h_decayAmps  = new complexT[nmbDecayAmps];
		const unsigned int decayAmpDimOld[3] = {_nmbEvents, 2, max(_nmbWavesRefl[0], _nmbWavesRefl[1])};
		const unsigned int decayAmpDimNew[3] = {2, max(_nmbWavesRefl[0], _nmbWavesRefl[1]), _nmbEvents};
		for (unsigned int iRefl = 0; iRefl < 2; ++iRefl)
			for (unsigned int iWave = 0; iWave < _nmbWavesRefl[iRefl]; ++iWave)
				for (unsigned int iEvt = 0; iEvt < _nmbEvents; ++iEvt) {
					const unsigned int decayAmpIndicesOld[3] = {iEvt, iRefl, iWave};
					const unsigned int offsetOld             = indicesToOffset<unsigned int>
						                                           (decayAmpIndicesOld, decayAmpDimOld, 3);
					const unsigned int decayAmpIndicesNew[3] = {iRefl, iWave, iEvt};
					const unsigned int offsetNew             = indicesToOffset<unsigned int>
						                                           (decayAmpIndicesNew, decayAmpDimNew, 3);
					h_decayAmps[offsetNew].real() = real(decayAmps[offsetOld]);
					h_decayAmps[offsetNew].imag() = imag(decayAmps[offsetOld]);
				}
	}

	// copy decay amps to device memory
	const unsigned int size = nmbDecayAmps * sizeof(complexT);
	cutilSafeCall(hipMalloc((void**)&_d_decayAmps, size));
	cutilSafeCall(hipMemcpy(_d_decayAmps, (reshuffleArray) ? h_decayAmps : decayAmps,
	                         size, hipMemcpyHostToDevice));
	printInfo << availableDeviceMem() / (1024. * 1024.) << " MiBytes left on CUDA device after loading "
	          << "decay amplitudes" << endl;
	return true;
}


template<typename complexT>
likelihoodInterface<complexT>::value_type
likelihoodInterface<complexT>::logLikelihood
(const complexT*    prodAmps,     // array of production amplitudes; [iRank][iRefl][iWave]
 const unsigned int nmbProdAmps,  // number of elements in production amplitude array
 const value_type   prodAmpFlat,  // (real) amplitude of flat wave
 const unsigned int rank)         // rank of spin-density matrix
{
	if (not prodAmps) {
		printErr << "null pointer to production amplitudes. aborting." << endl;
		throw;
	}

	// copy production amplitudes to device
	complexT* d_prodAmps;
	{
		const unsigned int size = nmbProdAmps * sizeof(complexT);
		cutilSafeCall(hipMalloc((void**)&d_prodAmps, size));
		cutilSafeCall(hipMemcpy(d_prodAmps, prodAmps, size, hipMemcpyHostToDevice));
	}

	// first summation stage
	value_type*        d_logLikelihoods0;
	const unsigned int nmbElements0 = _nmbThreadsPerBlock * _nmbBlocks;
	cutilSafeCall(hipMalloc((void**)&d_logLikelihoods0, sizeof(value_type) * nmbElements0));
	logLikelihoodKernel<complexT><<<_nmbBlocks, _nmbThreadsPerBlock>>>
		(d_prodAmps, prodAmpFlat * prodAmpFlat, _d_decayAmps, _nmbEvents, rank,
		 _nmbWavesRefl[0], _nmbWavesRefl[1], max(_nmbWavesRefl[0], _nmbWavesRefl[1]),
		 d_logLikelihoods0);
	//cutilSafeCall(hipDeviceSynchronize());
	// second summation stage
	value_type*        d_logLikelihoods1;
	const unsigned int nmbElements1 = _nmbThreadsPerBlock;
	cutilSafeCall(hipMalloc((void**)&d_logLikelihoods1, sizeof(value_type) * nmbElements1));
	sumKernel<value_type><<<1, _nmbThreadsPerBlock>>>(d_logLikelihoods0, nmbElements0,
	                                                  d_logLikelihoods1);
	//cutilSafeCall(hipDeviceSynchronize());
	// third and last summation stage
	value_type* d_logLikelihoods2;
	cutilSafeCall(hipMalloc((void**)&d_logLikelihoods2, sizeof(value_type)));
	sumKernel<value_type><<<1, 1>>>(d_logLikelihoods1, nmbElements1, d_logLikelihoods2);
	//cutilSafeCall(hipDeviceSynchronize());
	// copy result to host
	value_type logLikelihood;
	cutilSafeCall(hipMemcpy(&logLikelihood, d_logLikelihoods2,
	                         sizeof(value_type), hipMemcpyDeviceToHost));
	// cleanup
	cutilSafeCall(hipFree(d_prodAmps       ));
	cutilSafeCall(hipFree(d_logLikelihoods0));
	cutilSafeCall(hipFree(d_logLikelihoods1));
	cutilSafeCall(hipFree(d_logLikelihoods2));

	return logLikelihood;
}


//!!! this function does not work properly when -O3 is used in compilation
//!!!
//!!! the symptom is that from the _second_ invocation on, the function
//!!! calculates derivatives[0][0][0] wrongly. the reason seems to be
//!!! that the CUDA kernel is called with the iRank, iRefl, iWave from
//!!! the last kernel call of the previous function call;
//!!! testLikelihoodMockup.cc has a corresponding test case
//!!!
//!!! system specs: Ubuntu 10.04.1 LTS x86_64, CUDA 3.1, gcc version 4.3.4 (Ubuntu 4.3.4-10ubuntu1)
template<typename complexT>
likelihoodInterface<complexT>::value_type
likelihoodInterface<complexT>::logLikelihoodDeriv
(const complexT*    prodAmps,        // array of production amplitudes; [iRank][iRefl][iWave]
 const unsigned int nmbProdAmps,     // number of elements in production amplitude array
 const value_type   prodAmpFlat,     // (real) amplitude of flat wave
 const unsigned int rank,            // rank of spin-density matrix
 complexT*          derivatives,     // array of log likelihood derivatives; [iRank][iRefl][iWave]
 value_type&        derivativeFlat)  // log likelihood derivative of flat wave
{
	if (not prodAmps) {
		printErr << "null pointer to production amplitudes. aborting." << endl;
		throw;
	}

	// copy production amplitudes to device
	complexT* d_prodAmps;
	{
		const unsigned int size = nmbProdAmps * sizeof(complexT);
		cutilSafeCall(hipMalloc((void**)&d_prodAmps, size));
		cutilSafeCall(hipMemcpy(d_prodAmps, prodAmps, size, hipMemcpyHostToDevice));
	}

	// first stage: precalculate derivative term and likelihoods for each event
	complexT*   d_derivTerms;
	value_type* d_likelihoods;
	{
		cutilSafeCall(hipMalloc((void**)&d_derivTerms,  sizeof(complexT) * rank * 2 * _nmbEvents));
		cutilSafeCall(hipMalloc((void**)&d_likelihoods, sizeof(value_type) * _nmbEvents));
		//!!! this logic does not handle well smaller number of events; in
		//!!! this case number of threads per block should be chosen much smaller
		const unsigned int nmbBlocks = _nmbEvents / _nmbThreadsPerBlock + 1;
		//printInfo << "nmbEvents = " << _nmbEvents << ", nmbBlocks = " << nmbBlocks << endl;
		logLikelihoodDerivFirstTermKernel<complexT><<<nmbBlocks, _nmbThreadsPerBlock>>>
			(d_prodAmps, prodAmpFlat * prodAmpFlat, _d_decayAmps, _nmbEvents, rank,
			 _nmbWavesRefl[0], _nmbWavesRefl[1], max(_nmbWavesRefl[0], _nmbWavesRefl[1]),
			 d_derivTerms, d_likelihoods);
		//cutilSafeCall(hipDeviceSynchronize());
	}

	// second stage: calculate derivative  sums for all production amplitudes
	complexT*          d_derivatives;
	const unsigned int derivativeDim[3] = {rank, 2, max(_nmbWavesRefl[0], _nmbWavesRefl[1])};
	const unsigned int nmbDerivElements = derivativeDim[0] * derivativeDim[1] * derivativeDim[2];
	cutilSafeCall(hipMalloc((void**)&d_derivatives, sizeof(complexT) * nmbDerivElements));
	// cout << "!!!HERE1 d_derivatives = " << d_derivatives << endl;
	for (unsigned int iRank = 0; iRank < rank; ++iRank)
		for (unsigned int iRefl = 0; iRefl < 2; ++iRefl)
			for (unsigned int iWave = 0; iWave < _nmbWavesRefl[iRefl]; ++iWave) {
				// first summation stage
				complexT*          d_derivativeSums0;
				const unsigned int nmbElements0 = _nmbThreadsPerBlock * _nmbBlocks;
				cutilSafeCall(hipMalloc((void**)&d_derivativeSums0, sizeof(complexT) * nmbElements0));
				// cout << "!!!HERE1 [" << iRank << "][" << iRefl << "][" << iWave << "]" << endl;
				logLikelihoodDerivKernel<complexT><<<_nmbBlocks, _nmbThreadsPerBlock>>>
					(_d_decayAmps, d_derivTerms, d_likelihoods, _nmbEvents, rank,
					 max(_nmbWavesRefl[0], _nmbWavesRefl[1]), iRank, iRefl, iWave,
					 d_derivativeSums0);
				//cutilSafeCall(hipDeviceSynchronize());
				// second summation stage
				complexT*          d_derivativeSums1;
				const unsigned int nmbElements1 = _nmbThreadsPerBlock;
				cutilSafeCall(hipMalloc((void**)&d_derivativeSums1, sizeof(complexT) * nmbElements1));
				sumKernel<complexT><<<1, _nmbThreadsPerBlock>>>(d_derivativeSums0, nmbElements0,
				                                                d_derivativeSums1);
				//cutilSafeCall(hipDeviceSynchronize());
				cutilSafeCall(hipFree(d_derivativeSums0));
				// third and last summation stage
				const unsigned int derivativeIndices[3] = {iRank, iRefl, iWave};
				unsigned int offset                     = indicesToOffset<unsigned int>(derivativeIndices,
					                                                                      derivativeDim, 3);
				sumToMemCellKernel<complexT><<<1, 1>>>(d_derivativeSums1, nmbElements1,
				                                       d_derivatives, offset);
				//cutilSafeCall(hipDeviceSynchronize());
				// cutilSafeCall(hipFree(d_derivativeSums1));

				// cutilSafeCall(hipMemcpy(derivatives, d_derivatives,
				//                          sizeof(complexT) * nmbDerivElements, hipMemcpyDeviceToHost));
				// cout << "!!!HERE2 [" << iRank << "][" << iRefl << "][" << iWave << "]; [0] = "
				//      << "(" << maxPrecision(derivatives[0].real()) << ", "
				//      <<        maxPrecision(derivatives[0].imag()) << ")" << endl;
			}
	// copy result to host and cleanup
	cutilSafeCall(hipMemcpy(derivatives, d_derivatives,
	                         sizeof(complexT) * nmbDerivElements, hipMemcpyDeviceToHost));
	cutilSafeCall(hipFree(d_derivatives));
	// cout << "!!!HERE2 derivatives[0] = "
	//      << "(" << maxPrecision(derivatives[0].real()) << ", "
	//      <<        maxPrecision(derivatives[0].imag()) << ")" << endl;
	
	// flat wave requires special treatment	
	{
		// first summation stage
		value_type*        d_derivativeSumsFlat0;
		const unsigned int nmbElements0 = _nmbThreadsPerBlock * _nmbBlocks;
		cutilSafeCall(hipMalloc((void**)&d_derivativeSumsFlat0, sizeof(value_type) * nmbElements0));
		logLikelihoodDerivFlatKernel<complexT><<<_nmbBlocks, _nmbThreadsPerBlock>>>
			(prodAmpFlat, d_likelihoods, _nmbEvents, d_derivativeSumsFlat0);
		//cutilSafeCall(hipDeviceSynchronize());
		// second summation stage
		value_type*        d_derivativeSumsFlat1;
		const unsigned int nmbElements1 = _nmbThreadsPerBlock;
		cutilSafeCall(hipMalloc((void**)&d_derivativeSumsFlat1, sizeof(value_type) * nmbElements1));
		sumKernel<value_type><<<1, _nmbThreadsPerBlock>>>(d_derivativeSumsFlat0, nmbElements0,
		                                                  d_derivativeSumsFlat1);
		//cutilSafeCall(hipDeviceSynchronize());
		// third and last summation stage
		value_type* d_derivativeSumsFlat2;
		cutilSafeCall(hipMalloc((void**)&d_derivativeSumsFlat2, sizeof(value_type)));
		sumKernel<value_type><<<1, 1>>>(d_derivativeSumsFlat1, nmbElements1, d_derivativeSumsFlat2);
		//cutilSafeCall(hipDeviceSynchronize());
		// copy result to host
		cutilSafeCall(hipMemcpy(&derivativeFlat, d_derivativeSumsFlat2,
		                         sizeof(value_type), hipMemcpyDeviceToHost));
		// cleanup
		cutilSafeCall(hipFree(d_derivativeSumsFlat0));
		cutilSafeCall(hipFree(d_derivativeSumsFlat1));
		cutilSafeCall(hipFree(d_derivativeSumsFlat2));
	}

	// cleanup
	cutilSafeCall(hipFree(d_derivTerms ));
	cutilSafeCall(hipFree(d_likelihoods));
	cutilSafeCall(hipFree(d_prodAmps   ));

	return 0;
}


template<typename complexT>
ostream&
likelihoodInterface<complexT>::print(ostream& out)
{
  const unsigned int nGpuArchCoresPerSM[] = {1, 8, 32};  // from SDK/shared/inc/shrUtils.h

  if (not _cudaInitialized) {
	  printWarn << "CUDA device is not initialized." << endl;
	  return out;
  }
  
  // fields for both major & minor fields are 9999, if no CUDA capable devices are present
  if ((_cudaDeviceProp.major == 9999) and (_cudaDeviceProp.minor == 9999)) {
	  printWarn << "there is no CUDA device with ID " << _cudaDeviceId << endl;
	  return out;
  }
  out << "CUDA device[" << _cudaDeviceId << "]: '" << _cudaDeviceProp.name << "' properties:" << endl;
    
  // print info
  int driverVersion = 0;
  cutilSafeCall(hipDriverGetVersion(&driverVersion));
  int runtimeVersion = 0;     
  cutilSafeCall(hipRuntimeGetVersion(&runtimeVersion));
  out << "    driver version: .................................. " << driverVersion / 1000 << "." << driverVersion % 100 << endl
      << "    runtime version: ................................. " << runtimeVersion / 1000 << "." << runtimeVersion % 100 << endl
      << "    capability major revision number: ................ " << _cudaDeviceProp.major << endl
      << "    capability minor revision number: ................ " << _cudaDeviceProp.minor << endl
      << "    GPU clock frequency: ............................. " << _cudaDeviceProp.clockRate * 1e-6f << " GHz" << endl
      << "    number of multiprocessors: ....................... " << _cudaDeviceProp.multiProcessorCount << endl
      << "    number of cores: ................................. " << nGpuArchCoresPerSM[_cudaDeviceProp.major] * _cudaDeviceProp.multiProcessorCount << endl
      << "    warp size: ....................................... " << _cudaDeviceProp.warpSize << endl
      << "    maximum number of threads per block: ............. " << _cudaDeviceProp.maxThreadsPerBlock << endl
      << "    maximum block dimensions: ........................ " << _cudaDeviceProp.maxThreadsDim[0] << " x " << _cudaDeviceProp.maxThreadsDim[1]
                                                                   << " x " << _cudaDeviceProp.maxThreadsDim[2] << endl
      << "    maximum grid dimension ........................... " << _cudaDeviceProp.maxGridSize[0] << " x " << _cudaDeviceProp.maxGridSize[1]
                                                                   << " x " << _cudaDeviceProp.maxGridSize[2] << endl
      << "    total amount of global memory: ................... " << _cudaDeviceProp.totalGlobalMem / (1024. * 1024.) << " MiBytes" << endl
      << "    amount of available global memory: ............... " << availableDeviceMem() / (1024. * 1024.) << " MiBytes" << endl
      << "    total amount of constant memory: ................. " << _cudaDeviceProp.totalConstMem << " bytes" << endl 
      << "    total amount of shared memory per block: ......... " << _cudaDeviceProp.sharedMemPerBlock << " bytes" << endl
      << "    total number of registers available per block: ... " << _cudaDeviceProp.regsPerBlock << endl
      << "    maximum memory pitch: ............................ " << _cudaDeviceProp.memPitch << " bytes" << endl
      << "    texture alignment: ............................... " << _cudaDeviceProp.textureAlignment << " bytes" << endl
      << "    concurrent copy and execution: ................... " << ((_cudaDeviceProp.deviceOverlap)            ? "yes" : "no") << endl
      << "    run time limit on kernels: ....................... " << ((_cudaDeviceProp.kernelExecTimeoutEnabled) ? "yes" : "no") << endl
      << "    integrated: ...................................... " << ((_cudaDeviceProp.integrated)               ? "yes" : "no") << endl
      << "    support for host page-locked memory mapping: ..... " << ((_cudaDeviceProp.canMapHostMemory)         ? "yes" : "no") << endl
      << "    compute mode: .................................... ";
  if (_cudaDeviceProp.computeMode == hipComputeModeDefault)
	  out << "default (multiple host threads can use this device simultaneously)";
  else if (_cudaDeviceProp.computeMode == hipComputeModeExclusive)
	  out << "exclusive (only one host thread at a time can use this device)";
  else if (_cudaDeviceProp.computeMode == hipComputeModeProhibited)
	  out << "prohibited (no host thread can use this device)";
  else
	  out << "unknown";
  out << endl;
  return out;
}


// explicit specializations
template class likelihoodInterface<cuda::complex<float > >;
template class likelihoodInterface<cuda::complex<double> >;
