#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
//
//    Copyright 2010
//
//    This file is part of rootpwa
//
//    rootpwa is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    rootpwa is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with rootpwa. If not, see <http://www.gnu.org/licenses/>.
//
///////////////////////////////////////////////////////////////////////////
//-------------------------------------------------------------------------
// File and Version Information:
// $Rev::                             $: revision of last commit
// $Author::                          $: author of last commit
// $Date::                            $: date of last commit
//
// Description:
//      CUDA kernels for summation of large arrays and for likelihood
//      calculation
//
//
// Author List:
//      Philipp Meyer          TUM            (original author)
//
//
//-------------------------------------------------------------------------



namespace rpwa {

	namespace cuda {


		// generic cascadable kernel that computes sum of N values in an array
		template<typename T>
		__global__
		void
		sumKernel
		(const T*           d_valArray,  // array with values to sum up
		 const unsigned int nmbVals,     // total number of values all kernels have to process
		 T*                 d_sumArray)  // output array of partial sums with one entry for each kernel
		{
			const unsigned int threadId   = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int nmbThreads = gridDim.x * blockDim.x;
  
			// #pragma unroll 16
			T sum = 0;
			for (unsigned int i = threadId; i < nmbVals; i += nmbThreads)
				sum += d_valArray[i];

			d_sumArray[threadId] = sum;
		}


		// kernel that calculates real-data term of log likelihood sum for a number of events
		template<typename complexT>
		__global__
		void
		logLikelihoodKernel
		(const complexT*                     d_prodAmps,        // 3-dim. array of production amplitudes [iRank][iRefl][iWave]
		 const typename complexT::value_type prodAmpFlat2,      // squared production amplitude of flat wave
		 const complexT*                     d_decayAmps,       // 3-dim. array of decay amplitudes [iRefl][iWave][iEvt]
		 const unsigned int                  nmbEvents,         // total number of events all kernels have to process
		 const unsigned int                  rank,              // rank of spin-density matrix
		 const unsigned int                  nmbWavesReflNeg,   // number waves with negative reflectivity
		 const unsigned int                  nmbWavesReflPos,   // number waves with positive reflectivity
		 const unsigned int                  nmbWavesMax,       // maximum extent of iWave index for production and decay amplitude arrays
		 typename complexT::value_type*      d_logLikelihoods)  // output array of partial log likelihood sums with one entry for each kernel
		{
			const unsigned int threadId   = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int nmbThreads = gridDim.x * blockDim.x;

			const unsigned int nmbWavesRefl[2] = {nmbWavesReflNeg, nmbWavesReflPos};
			// define extents of arrays
			const unsigned int prodAmpDim [3] = {rank, 2,           nmbWavesMax};
			const unsigned int decayAmpDim[3] = {2,    nmbWavesMax, nmbEvents};
			// loop over events and calculate real-data term of log likelihood
			typename complexT::value_type logLikelihood = 0;
			for (unsigned int iEvt = threadId; iEvt < nmbEvents; iEvt += nmbThreads) {
				typename complexT::value_type likelihood = 0;  // likelihood for this event
				for (unsigned int iRank = 0; iRank < rank; ++iRank) {  // incoherent sum over ranks
					for (unsigned int iRefl = 0; iRefl < 2; ++iRefl) {  // incoherent sum over reflectivities
						complexT ampProdSum = 0;  // amplitude sum for negative/positive reflectivity for this rank
						for (unsigned int iWave = 0; iWave < nmbWavesRefl[iRefl]; ++iWave) {  // coherent sum over waves
							// compute likelihood term
							const unsigned int prodAmpIndices [3] = {iRank, iRefl, iWave};
							const unsigned int decayAmpIndices[3] = {iRefl, iWave, iEvt};
							ampProdSum +=
								  d_prodAmps [indicesToOffset<unsigned int>(prodAmpIndices,  prodAmpDim,  3)]
								* d_decayAmps[indicesToOffset<unsigned int>(decayAmpIndices, decayAmpDim, 3)];
						}
						likelihood += norm(ampProdSum);
					}
				}
				likelihood    += prodAmpFlat2;
				logLikelihood -= log(likelihood);  // accumulate log likelihood
			}
			// write result
			d_logLikelihoods[threadId] = logLikelihood;
		}


		// kernel that calculates the first real-data term of derivative of log likelihood sum
		// that is independent from the derivative wave index for single event
		template<typename complexT>
		__global__
		void
		logLikelihoodDerivTerm1Kernel
		(const complexT*    d_prodAmps,       // 3-dim. array of production amplitudes [iRank][iRefl][iWave]
		 const complexT*    d_decayAmps,      // 3-dim. array of decay amplitudes [iRefl][iWave][iEvt]
		 const unsigned int nmbEvents,        // total number of events all kernels have to process
		 const unsigned int rank,             // rank of spin-density matrix
		 const unsigned int nmbWavesReflNeg,  // number waves with negative reflectivity
		 const unsigned int nmbWavesReflPos,  // number waves with positive reflectivity
		 const unsigned int nmbWavesMax,      // maximum extent of iWave index for production and decay amplitude arrays
		 complexT*          d_derivTerm1)     // 3-dim. output array of first derivative term [iRank][iRefl][iEvt]
		{
			const unsigned int iEvt = blockIdx.x * blockDim.x + threadIdx.x;
			if (iEvt < nmbEvents) {
				const unsigned int nmbWavesRefl[2] = {nmbWavesReflNeg, nmbWavesReflPos};
				// define extents of arrays
				const unsigned int prodAmpDim   [3] = {rank, 2,           nmbWavesMax};
				const unsigned int decayAmpDim  [3] = {2,    nmbWavesMax, nmbEvents};
				const unsigned int derivTerm1Dim[3] = {rank, 2,           nmbEvents};
				for (unsigned int iRank = 0; iRank < rank; ++iRank) {  // incoherent sum over ranks
					for (unsigned int iRefl = 0; iRefl < 2; ++iRefl) {  // incoherent sum over reflectivities
						complexT ampProdSum = 0;  // amplitude sum for negative/positive reflectivity for this rank
						for (unsigned int iWave = 0; iWave < nmbWavesRefl[iRefl]; ++iWave) {  // coherent sum over waves
							const unsigned int prodAmpIndices [3] = {iRank, iRefl, iWave};
							const unsigned int decayAmpIndices[3] = {iRefl, iWave, iEvt};
							ampProdSum +=
								  d_prodAmps [indicesToOffset<unsigned int>(prodAmpIndices,  prodAmpDim,  3)]
								* d_decayAmps[indicesToOffset<unsigned int>(decayAmpIndices, decayAmpDim, 3)];
						}
						// write result
						const unsigned int derivTerm1Indices[3] = {iRank, iRefl, iEvt};
						d_derivTerm1[indicesToOffset<unsigned int>(derivTerm1Indices, derivTerm1Dim, 3)]
							= ampProdSum;
					}
				}
			}
		}


		// kernel that operates on output of logLikelihoodDerivTerm1Kernel and 
		// calculates the first real-data term of derivative of log likelihood sum
		// that is independent from the derivative wave index for single event
		template<typename complexT>
		__global__
		void
		logLikelihoodDerivKernel
		(const typename complexT::value_type prodAmpFlat2,      // squared production amplitude of flat wave
		 const complexT*                     d_decayAmps,       // 3-dim. array of decay amplitudes [iRefl][iWave][iEvt]
		 const complexT*                     d_derivTerm1,      // precalculated 3-dim. array of first derivative term [iRank][iRefl][iEvt]
		 const unsigned int                  nmbEvents,         // total number of events all kernels have to process
		 const unsigned int                  rank,              // rank of spin-density matrix
		 const unsigned int                  nmbWavesMax,       // maximum extent of iWave index for production and decay amplitude arrays
		 const unsigned int                  jRank,
		 const unsigned int                  jRefl,
		 const unsigned int                  jWave,
		 complexT*                           d_derivativeSums)  // output array of partial derivative sums with one entry for each kernel
		{
			const unsigned int threadId   = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int nmbThreads = gridDim.x * blockDim.x;

			// define extents of arrays
			const unsigned int decayAmpDim[3]   = {2,    nmbWavesMax, nmbEvents};
			const unsigned int derivTerm1Dim[3] = {rank, 2,           nmbEvents};
			// loop over events and calculate real-data term of derivative of log likelihood
			complexT derivativeSum = 0;
			for (unsigned int iEvt = threadId; iEvt < nmbEvents; iEvt += nmbThreads) {
				// calculate likelihood of event
				typename complexT::value_type likelihood = 0;  // likelihood for this event
				for (unsigned int iRank = 0; iRank < rank; ++iRank) {  // incoherent sum over ranks
					for (unsigned int iRefl = 0; iRefl < 2; ++iRefl) {  // incoherent sum over reflectivities
						const unsigned int derivTerm1Indices[3] = {iRank, iRefl, iEvt};
						likelihood += norm
							(d_derivTerm1[indicesToOffset<unsigned int>(derivTerm1Indices, derivTerm1Dim, 3)]);
					}
				}
				likelihood += prodAmpFlat2;
				// multiply derivative term 1 with with complex conjugate of
				// decay amplitude of the wave with the derivative wave index
				const unsigned int derivTerm1Indices[3] = {jRank, jRefl, iEvt};
				const unsigned int decayAmpIndices[3]   = {jRefl, jWave, iEvt};
				const complexT derivative =
					  d_derivTerm1[indicesToOffset<unsigned int>(derivTerm1Indices, derivTerm1Dim, 3)]
					* conj(d_decayAmps [indicesToOffset<unsigned int>(decayAmpIndices, decayAmpDim, 3)]);
				
				// incorporate factor from derivative of log
				const typename complexT::value_type factor = 2. / likelihood;
				derivativeSum -= factor * derivative;
			}
			// write result
			d_derivativeSums[threadId] = derivativeSum;
		}


	}  // namespace cuda

}  // namespace rpwa
