#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
//
//    Copyright 2010
//
//    This file is part of rootpwa
//
//    rootpwa is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    rootpwa is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with rootpwa. If not, see <http://www.gnu.org/licenses/>.
//
///////////////////////////////////////////////////////////////////////////
//-------------------------------------------------------------------------
// File and Version Information:
// $Rev::                             $: revision of last commit
// $Author::                          $: author of last commit
// $Date::                            $: date of last commit
//
// Description:
//      CUDA kernels for summation of large arrays and for likelihood
//      calculation
//
//
// Author List:
//      Philipp Meyer          TUM            (original author)
//
//
//-------------------------------------------------------------------------


namespace rpwa {

	namespace cuda {


		// generic cascadable kernel that computes sum of N values in an array
		template<typename T>
		__global__
		void
		sumKernel
		(const T*           d_valArray,  // array with values to sum up
		 const unsigned int nmbVals,     // total number of values all kernels have to process
		 T*                 d_sumArray)  // output array of partial sums with one entry for each kernel
		{
			const unsigned int threadId   = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int nmbThreads = gridDim.x * blockDim.x;
  
			// #pragma unroll 16
			T sum = T(0);
			for (unsigned int i = threadId; i < nmbVals; i += nmbThreads)
				sum += d_valArray[i];

			d_sumArray[threadId] = sum;
		}


		// generic kernel that computes sum of N values in an array and
		// writes result to memory location defined by pointer and offset
		template<typename T>
		__global__
		void
		sumToMemCellKernel
		(const T*           d_valArray,    // array with values to sum up
		 const unsigned int nmbVals,       // total number of values all kernels have to process
		 T*                 d_sumArray,    // output array of partial sums
		 const unsigned int outputOffset)  // offset of memory location the result will be written to
		{
			const unsigned int threadId   = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int nmbThreads = gridDim.x * blockDim.x;
  
			// #pragma unroll 16
			T sum = T(0);
			for (unsigned int i = threadId; i < nmbVals; i += nmbThreads)
				sum += d_valArray[i];

			d_sumArray[outputOffset] = sum;
		}


		// kernel that calculates real-data term of log likelihood sum for a number of events
		template<typename complexT>
		__global__
		void
		logLikelihoodKernel
		(const complexT*                     d_prodAmps,        // 3-dim. array of production amplitudes; [iRank][iRefl][iWave]
		 const typename complexT::value_type prodAmpFlat2,      // squared production amplitude of flat wave
		 const complexT*                     d_decayAmps,       // 3-dim. array of decay amplitudes; [iRefl][iWave][iEvt]
		 const unsigned int                  nmbEvents,         // total number of events all kernels have to process
		 const unsigned int                  rank,              // rank of spin-density matrix
		 const unsigned int                  nmbWavesReflNeg,   // number waves with negative reflectivity
		 const unsigned int                  nmbWavesReflPos,   // number waves with positive reflectivity
		 const unsigned int                  nmbWavesMax,       // maximum extent of iWave index for production and decay amplitude arrays
		 typename complexT::value_type*      d_logLikelihoods)  // output array of partial log likelihood sums with one entry for each kernel
		{
			const unsigned int threadId   = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int nmbThreads = gridDim.x * blockDim.x;

			const unsigned int nmbWavesRefl[2] = {nmbWavesReflNeg, nmbWavesReflPos};
			// define extents of arrays
			const unsigned int prodAmpDim [3] = {rank, 2,           nmbWavesMax};
			const unsigned int decayAmpDim[3] = {2,    nmbWavesMax, nmbEvents};
			// loop over events and calculate real-data term of log likelihood
			typename complexT::value_type logLikelihood = 0;
			for (unsigned int iEvt = threadId; iEvt < nmbEvents; iEvt += nmbThreads) {
				typename complexT::value_type likelihood = 0;  // likelihood for this event
				for (unsigned int iRank = 0; iRank < rank; ++iRank) {  // incoherent sum over ranks
					for (unsigned int iRefl = 0; iRefl < 2; ++iRefl) {  // incoherent sum over reflectivities
						complexT ampProdSum = complexT(0);  // amplitude sum for negative/positive reflectivity for this rank
						for (unsigned int iWave = 0; iWave < nmbWavesRefl[iRefl]; ++iWave) {  // coherent sum over waves
							// compute likelihood term
							const unsigned int prodAmpIndices [3] = {iRank, iRefl, iWave};
							const unsigned int decayAmpIndices[3] = {iRefl, iWave, iEvt};
							ampProdSum +=
								  d_prodAmps [indicesToOffset<unsigned int>(prodAmpIndices,  prodAmpDim,  3)]
								* d_decayAmps[indicesToOffset<unsigned int>(decayAmpIndices, decayAmpDim, 3)];
						}
						likelihood += norm(ampProdSum);
					}
				}
				likelihood    += prodAmpFlat2;
				logLikelihood -= log(likelihood);  // accumulate log likelihood
			}
			// write result
			d_logLikelihoods[threadId] = logLikelihood;
		}


		// kernel that calculates the real-data term of the derivative of
		// the log likelihood that is independent of the indices of the
		// production amplitude w.r.t. which the derivative is taken
		template<typename complexT>
		__global__
		void
		logLikelihoodDerivFirstTermKernel
		(const complexT*                     d_prodAmps,       // 3-dim. array of production amplitudes; [iRank][iRefl][iWave]
		 const typename complexT::value_type prodAmpFlat2,     // squared production amplitude of flat wave
		 const complexT*                     d_decayAmps,      // 3-dim. array of decay amplitudes; [iRefl][iWave][iEvt]
		 const unsigned int                  nmbEvents,        // total number of events all kernels have to process
		 const unsigned int                  rank,             // rank of spin-density matrix
		 const unsigned int                  nmbWavesReflNeg,  // number waves with negative reflectivity
		 const unsigned int                  nmbWavesReflPos,  // number waves with positive reflectivity
		 const unsigned int                  nmbWavesMax,      // maximum extent of iWave index for production and decay amplitude arrays
		 complexT*                           d_derivTerms,     // 3-dim. output array of first derivative terms; [iRank][iRefl][iEvt]
		 typename complexT::value_type*      d_likelihoods)    // output array of likelihoods for each event
		{
			const unsigned int iEvt = blockIdx.x * blockDim.x + threadIdx.x;
			if (iEvt < nmbEvents) {
				const unsigned int nmbWavesRefl[2] = {nmbWavesReflNeg, nmbWavesReflPos};
				// define extents of arrays
				const unsigned int            prodAmpDim  [3] = {rank, 2,           nmbWavesMax};
				const unsigned int            decayAmpDim [3] = {2,    nmbWavesMax, nmbEvents};
				const unsigned int            derivTermDim[3] = {rank, 2,           nmbEvents};
				typename complexT::value_type likelihood      = 0;  // likelihood for this event
				for (unsigned int iRank = 0; iRank < rank; ++iRank) {  // incoherent sum over ranks
					for (unsigned int iRefl = 0; iRefl < 2; ++iRefl) {  // incoherent sum over reflectivities
						complexT ampProdSum = complexT(0);  // amplitude sum for negative/positive reflectivity for this rank
						for (unsigned int iWave = 0; iWave < nmbWavesRefl[iRefl]; ++iWave) {  // coherent sum over waves
							const unsigned int prodAmpIndices [3] = {iRank, iRefl, iWave};
							const unsigned int decayAmpIndices[3] = {iRefl, iWave, iEvt};
							ampProdSum +=
								  d_prodAmps [indicesToOffset<unsigned int>(prodAmpIndices,  prodAmpDim,  3)]
								* d_decayAmps[indicesToOffset<unsigned int>(decayAmpIndices, decayAmpDim, 3)];
						}
						likelihood += norm(ampProdSum);
						// write derivative term
						const unsigned int derivTermIndices[3] = {iRank, iRefl, iEvt};
						d_derivTerms[indicesToOffset<unsigned int>(derivTermIndices, derivTermDim, 3)]
							= ampProdSum;
					}
				}
				likelihood += prodAmpFlat2;
				// write likelihood
				d_likelihoods[iEvt] = likelihood;
			}
		}


		// kernel that operates on the output of logLikelihoodDerivFirstTermKernel
		// and calculates the real-data derivative sum of the log likelihood for a
		// particular production amplitude
		template<typename complexT>
		__global__
		void
		logLikelihoodDerivKernel
		(const complexT*                      d_decayAmps,       // 3-dim. array of decay amplitudes; [iRefl][iWave][iEvt]
		 const complexT*                      d_derivTerms,      // precalculated 3-dim. array of first derivative terms; [iRank][iRefl][iEvt]
		 const typename complexT::value_type* d_likelihoods,     // precalculated array of likelihoods for each event
		 const unsigned int                   nmbEvents,         // total number of events all kernels have to process
		 const unsigned int                   rank,              // rank of spin-density matrix
		 const unsigned int                   nmbWavesMax,       // maximum extent of iWave index for production and decay amplitude arrays
		 const unsigned int                   iRank,             // rank of production amplitude w.r.t. which derivative is taken
		 const unsigned int                   iRefl,             // reflectivity of production amplitude w.r.t. which derivative is taken
		 const unsigned int                   iWave,             // wave index of production amplitude w.r.t. which derivative is taken
		 complexT*                            d_derivativeSums)  // output array of partial derivative sums with one entry for each kernel
		{
			const unsigned int threadId   = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int nmbThreads = gridDim.x * blockDim.x;

			// define extents of arrays
			const unsigned int decayAmpDim [3] = {2,    nmbWavesMax, nmbEvents};
			const unsigned int derivTermDim[3] = {rank, 2,           nmbEvents};
			// loop over events and calculate real-data term of derivative of log likelihood
			complexT derivativeSum = complexT(0);
			for (unsigned int iEvt = threadId; iEvt < nmbEvents; iEvt += nmbThreads) {
				// multiply derivative term 1 with with complex conjugate of
				// decay amplitude of the wave with the derivative wave index
				const unsigned int decayAmpIndices [3] = {iRefl, iWave, iEvt};
				const unsigned int derivTermIndices[3] = {iRank, iRefl, iEvt};
				const complexT     derivative          =
					  d_derivTerms[indicesToOffset<unsigned int>(derivTermIndices, derivTermDim, 3)]
					* conj(d_decayAmps[indicesToOffset<unsigned int>(decayAmpIndices, decayAmpDim, 3)]);
				// apply factor from derivative of log
				derivativeSum -= (2. / d_likelihoods[iEvt]) * derivative;
			}
			// write result
			d_derivativeSums[threadId] = derivativeSum;
		}


		template<typename complexT>
		__global__
		void
		logLikelihoodDerivKernelXXX
		(const complexT*                      d_decayAmps,       // 3-dim. array of decay amplitudes; [iRefl][iWave][iEvt]
		 const complexT*                      d_derivTerms,      // precalculated 3-dim. array of first derivative terms; [iRank][iRefl][iEvt]
		 const typename complexT::value_type* d_likelihoods,     // precalculated array of likelihoods for each event
		 const unsigned int                   nmbEvents,         // total number of events all kernels have to process
		 const unsigned int                   rank,              // rank of spin-density matrix
		 const unsigned int                   nmbWavesReflNeg,   // number waves with negative reflectivity
		 const unsigned int                   nmbWavesReflPos,   // number waves with positive reflectivity
		 const unsigned int                   nmbWavesMax,       // maximum extent of iWave index for production and decay amplitude arrays
		 complexT*                            d_derivativeSums)  // output array of partial derivative sums with one entry for each kernel
		{
			const unsigned int threadId   = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int nmbThreads = gridDim.x * blockDim.x;

			const unsigned int nmbWavesRefl[2] = {nmbWavesReflNeg, nmbWavesReflPos};
			// define extents of arrays
			const unsigned int decayAmpDim [3] = {2,    nmbWavesMax, nmbEvents};
			const unsigned int derivTermDim[3] = {rank, 2,           nmbEvents};
			const unsigned int derivSumDim [4] = {rank, 2,           nmbWavesMax, nmbThreads};
			// loop over events and calculate real-data term of derivative of log likelihood
			for (unsigned int iRank = 0; iRank < rank; ++iRank)
				for (unsigned int iRefl = 0; iRefl < 2; ++iRefl)
					for (unsigned int iWave = 0; iWave < nmbWavesRefl[iRefl]; ++iWave) {
						complexT derivativeSum = complexT(0);
						for (unsigned int iEvt = threadId; iEvt < nmbEvents; iEvt += nmbThreads) {
							// multiply derivative term 1 with with complex conjugate of
							// decay amplitude of the wave with the derivative wave index
							const unsigned int decayAmpIndices [3] = {iRefl, iWave, iEvt};
							const unsigned int derivTermIndices[3] = {iRank, iRefl, iEvt};
							const complexT     derivative          =
								  d_derivTerms[indicesToOffset<unsigned int>(derivTermIndices, derivTermDim, 3)]
								* conj(d_decayAmps[indicesToOffset<unsigned int>(decayAmpIndices, decayAmpDim, 3)]);
							// apply factor from derivative of log
							derivativeSum -= (2. / d_likelihoods[iEvt]) * derivative;
						}
						// write result
						const unsigned int derivSumIndices[4] = {iRank, iRefl, iWave, threadId};
						d_derivativeSums[indicesToOffset<unsigned int>(derivSumIndices, derivSumDim, 4)]
							= derivativeSum;
					}
		}


		template<typename T>
		__global__
		void
		sumKernelXXX
		(const T*           d_derivatives,     // array with values to sum up
		 const unsigned int nmbDerivatives,    // total number of values all kernels have to process
		 const unsigned int rank,              // rank of spin-density matrix
		 const unsigned int nmbWavesReflNeg,   // number waves with negative reflectivity
		 const unsigned int nmbWavesReflPos,   // number waves with positive reflectivity
		 const unsigned int nmbWavesMax,       // maximum extent of iWave index for production and decay amplitude arrays
		 T*                 d_derivativeSums,  // output array of partial sums with one entry for each kernel
		 const unsigned int nmbSums)           // number of elements in output array
		{
			const unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
			if (threadId < nmbSums) {
				const unsigned int nmbWavesRefl[2] = {nmbWavesReflNeg, nmbWavesReflPos};
				// define extents of derivative array
				const unsigned int derivDim   [4] = {rank, 2, nmbWavesMax, nmbDerivatives};
				const unsigned int derivSumDim[4] = {rank, 2, nmbWavesMax, nmbSums};
				for (unsigned int iRank = 0; iRank < rank; ++iRank)
					for (unsigned int iRefl = 0; iRefl < 2; ++iRefl)
						for (unsigned int iWave = 0; iWave < nmbWavesRefl[iRefl]; ++iWave) {
							T derivativeSum = T(0);
							for (unsigned int i = threadId; i < nmbDerivatives; i += nmbSums) {
								const unsigned int derivIndices[4] = {iRank, iRefl, iWave, i};
								derivativeSum
									+= d_derivatives[indicesToOffset<unsigned int>(derivIndices, derivDim, 4)];
							}
							const unsigned int derivSumIndices[4] = {iRank, iRefl, iWave, threadId};
							d_derivativeSums[indicesToOffset<unsigned int>(derivSumIndices, derivSumDim, 4)]
								= derivativeSum;
						}
			}
		}


		// kernel that operates on the output of logLikelihoodDerivFirstTermKernel
		// and calculates the real-data derivative sum of the log likelihood for the flat wave
		template<typename complexT>
		__global__
		void
		logLikelihoodDerivFlatKernel
		(const typename complexT::value_type  prodAmpFlat,           // (real) production amplitude of flat wave
		 const typename complexT::value_type* d_likelihoods,         // precalculated array of likelihoods for each event
		 const unsigned int                   nmbEvents,             // total number of events all kernels have to process
		 typename complexT::value_type*       d_derivativeFlatSums)  // output array of partial derivative sums with one entry for each kernel
		{
			const unsigned int threadId   = blockIdx.x * blockDim.x + threadIdx.x;
			const unsigned int nmbThreads = gridDim.x * blockDim.x;

			// loop over events and calculate real-data term of derivative of log likelihood
			typename complexT::value_type derivativeFlatSum = 0;
			for (unsigned int iEvt = threadId; iEvt < nmbEvents; iEvt += nmbThreads)
				derivativeFlatSum -= (2. / d_likelihoods[iEvt]) * prodAmpFlat;
			// write result
			d_derivativeFlatSums[threadId] = derivativeFlatSum;
		}


	}  // namespace cuda

}  // namespace rpwa
